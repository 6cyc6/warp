#include "core.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

void* alloc_device(size_t s)
{
    void* ptr;
    check_cuda(hipMalloc(&ptr, s));

    return ptr;
}

void free_device(void* ptr)
{
    check_cuda(hipFree(ptr));
}

void memcpy_h2d(void* dest, void* src, size_t n)
{
    check_cuda(hipMemcpyAsync(dest, src, n, hipMemcpyHostToDevice));
}

void memcpy_d2h(void* dest, void* src, size_t n)
{
    check_cuda(hipMemcpyAsync(dest, src, n, hipMemcpyDeviceToHost));
}

void memcpy_d2d(void* dest, void* src, size_t n)
{
    check_cuda(hipMemcpyAsync(dest, src, n, hipMemcpyDeviceToDevice));
}

void memset_device(void* dest, int value, size_t n)
{
    check_cuda(hipMemsetAsync(dest, value, n));
}

void synchronize()
{
    check_cuda(hipStreamSynchronize(0));
}

void array_inner_device(uint64_t a, uint64_t b, uint64_t out, int len)
{

}

void array_sum_device(uint64_t a, uint64_t out, int len)
{
    
}


uint64_t cuda_check_device()
{
    hipDeviceSynchronize();
    return hipPeekAtLastError(); 
}


#if defined(__linux__)
#include <dlfcn.h>
static void* GetProcAddress(void* handle, const char* name) { return dlsym(handle, name); }
#endif

#if defined(_WIN32)
#include <windows.h>
#endif

typedef hipError_t CUDAAPI cuInit_t(unsigned int);
typedef hipError_t CUDAAPI cuDeviceGet_t(hipDevice_t *dev, int ordinal);
typedef hipError_t CUDAAPI cuCtxGetCurrent_t(hipCtx_t* ctx);
typedef hipError_t CUDAAPI cuCtxSetCurrent_t(hipCtx_t ctx);
typedef hipError_t CUDAAPI cuCtxCreate_t(hipCtx_t* pctx, unsigned int flags, hipDevice_t dev);
typedef hipError_t CUDAAPI cuCtxDestroy_t(hipCtx_t pctx);

static cuInit_t* cuInit_f;
static cuCtxGetCurrent_t* cuCtxGetCurrent_f;
static cuCtxSetCurrent_t* cuCtxSetCurrent_f;
//static cuCtxCreate_t* cuCtxCreate_f;
//static cuCtxDestroy_t* cuCtxDestroy_f;
//static cuDeviceGet_t* cuDeviceGet_f;

static hipCtx_t g_cuda_context;
static hipCtx_t g_save_context;

bool cuda_init()
{
    #if defined(_WIN32)
        static HMODULE hCudaDriver = LoadLibrary("nvcuda.dll");
    #elif defined(__linux__)
        static void* hCudaDriver = dlopen("libcuda.so", RTLD_NOW);
    #endif

    if (hCudaDriver == NULL)
        return false;

	cuInit_f = (cuInit_t*)GetProcAddress(hCudaDriver, "hipInit");
	cuCtxSetCurrent_f = (cuCtxSetCurrent_t*)GetProcAddress(hCudaDriver, "hipCtxSetCurrent");
	cuCtxGetCurrent_f = (cuCtxGetCurrent_t*)GetProcAddress(hCudaDriver, "hipCtxGetCurrent");
//	cuCtxCreate_f = (cuCtxCreate_t*)GetProcAddress(hCudaDriver, "hipCtxCreate");
//	cuCtxDestroy_f = (cuCtxDestroy_t*)GetProcAddress(hCudaDriver, "hipCtxDestroy");
//	cuDeviceGet_f = (cuDeviceGet_t*)GetProcAddress(hCudaDriver, "hipDeviceGet");

    if (cuInit_f == NULL)
        return false;

    if (cuInit_f(0) != hipSuccess)
		return false;

    hipCtx_t ctx;
    cuCtxGetCurrent_f(&ctx);

    if (ctx == NULL)
    {
        // create a new default runtime context
        hipSetDevice(0);
        cuCtxGetCurrent_f(&ctx);
    }
    
    g_cuda_context = ctx;
    
    return true;
}

void cuda_acquire_context()
{
    cuCtxGetCurrent_f(&g_save_context);
    cuCtxSetCurrent_f(g_cuda_context);
}

void cuda_restore_context()
{
    cuCtxSetCurrent_f(g_save_context);
}


void* cuda_get_context()
{
	hipCtx_t ctx;
	if (cuCtxGetCurrent_f(&ctx) == hipSuccess)
	    return ctx;
    else
        return NULL;
}

void cuda_set_context(void* ctx)
{
    cuCtxSetCurrent_f((hipCtx_t)ctx);
}

// impl. files
#include "bvh.cu"
#include "mesh.cu"
//#include "spline.inl"
//#include "volume.inl"


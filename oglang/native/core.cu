#include "core.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

void* alloc_device(size_t s)
{
    void* ptr;
    hipMalloc(&ptr, s);

    return ptr;
}

void free_device(void* ptr)
{
    hipFree(ptr);
}


void memcpy_h2d(void* dest, void* src, size_t n)
{
    hipMemcpyAsync(dest, src, n, hipMemcpyHostToDevice);
}

void memcpy_d2h(void* dest, void* src, size_t n)
{
    hipMemcpyAsync(dest, src, n, hipMemcpyDeviceToHost);
}

void memcpy_d2d(void* dest, void* src, size_t n)
{
    hipMemcpyAsync(dest, src, n, hipMemcpyDeviceToDevice);
}

void memset_device(void* dest, int value, size_t n)
{
    hipMemsetAsync(dest, value, n);
}

void synchronize()
{
    hipStreamSynchronize(0);
}

// impl. files
#include "bvh.cu"
#include "mesh.cu"
//#include "spline.inl"
//#include "volume.inl"


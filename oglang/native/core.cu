#include "core.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

void init()
{

}

void shutdown()
{

}

void* alloc_host(size_t s)
{
    return malloc(s);
}

void* alloc_device(size_t s)
{
    void* ptr;
    hipMalloc(&ptr, s);

    return ptr;
}

void free_host(void* ptr)
{
    free(ptr);
}

void free_device(void* ptr)
{
    hipFree(ptr);
}

void memcpy_h2h(void* dest, void* src, size_t n)
{
    memcpy(dest, src, n);
}

void memcpy_h2d(void* dest, void* src, size_t n)
{
    hipMemcpyAsync(dest, src, n, hipMemcpyHostToDevice);
}

void memcpy_d2h(void* dest, void* src, size_t n)
{
    hipMemcpyAsync(dest, src, n, hipMemcpyDeviceToHost);
}

void memcpy_d2d(void* dest, void* src, size_t n)
{
    hipMemcpyAsync(dest, src, n, hipMemcpyDeviceToDevice);
}

void memset_host(void* dest, int value, size_t n)
{
    memset(dest, value, n);
}

void memset_device(void* dest, int value, size_t n)
{
    hipMemsetAsync(dest, value, n);
}

void synchronize()
{
    hipDeviceSynchronize();
}

// impl. files
#include "bvh.cu"
#include "mesh.cu"
//#include "spline.cuh"
//#include "volume.cuh"


#include "hip/hip_runtime.h"
#include "mesh.h"
#include "bvh.h"

using namespace og;

#include <map>

namespace
{
    // host-side copy of mesh descriptors, maps GPU mesh address (id) to a CPU desc
    std::map<uint64_t, Mesh> g_mesh_descriptors;


    bool get_descriptor(uint64_t id, Mesh& mesh)
    {
        const auto& iter = g_mesh_descriptors.find(id);
        if (iter == g_mesh_descriptors.end())
            return false;
        else
            mesh = iter->second;
            return true;
    }

    void add_descriptor(uint64_t id, const Mesh& mesh)
    {
        g_mesh_descriptors[id] = mesh;
    }

    void rem_descriptor(uint64_t id)
    {
        g_mesh_descriptors.erase(id);

    }

} // anonymous namespace




uint64_t mesh_create_host(vec3* points, int* indices, int num_points, int num_tris)
{
    Mesh* m = new Mesh();

    m->points = points;
    m->indices = indices;

    m->num_points = num_points;
    m->num_tris = num_tris;

    m->bounds = new bounds3[num_tris];

    for (int i=0; i < num_tris; ++i)
    {
        m->bounds[i].add_point(points[indices[i*3+0]]);
        m->bounds[i].add_point(points[indices[i*3+1]]);
        m->bounds[i].add_point(points[indices[i*3+2]]);
    }

    m->bvh = bvh_create(m->bounds, num_tris);

    return (uint64_t)m;
}

uint64_t mesh_create_device(vec3* points, int* indices, int num_points, int num_tris)
{
    Mesh mesh;

    mesh.points = points;
    mesh.indices = indices;

    mesh.num_points = num_points;
    mesh.num_tris = num_tris;

    {
        // todo: BVH creation only on CPU at the moment so temporarily bring all the data back to host
        vec3* points_host = (vec3*)alloc_host(sizeof(vec3)*num_points);
        int* indices_host = (int*)alloc_host(sizeof(int)*num_tris*3);
        bounds3* bounds_host = (bounds3*)alloc_host(sizeof(bounds3)*num_tris);

        memcpy_d2h(points_host, points, sizeof(vec3)*num_points);
        memcpy_d2h(indices_host, indices, sizeof(int)*num_tris*3);
        synchronize();

        for (int i=0; i < num_tris; ++i)
        {
            bounds_host[i].add_point(points_host[indices_host[i*3+0]]);
            bounds_host[i].add_point(points_host[indices_host[i*3+1]]);
            bounds_host[i].add_point(points_host[indices_host[i*3+2]]);
        }

        BVH bvh_host = bvh_create(bounds_host, num_tris);
        BVH bvh_device = bvh_clone(bvh_host);

        bvh_destroy_host(bvh_host);

        // save gpu-side copy of bounds
        mesh.bounds = (bounds3*)alloc_device(sizeof(bounds3)*num_tris);
        memcpy_h2d(mesh.bounds, bounds_host, sizeof(bounds3)*num_tris);

        free_host(points_host);
        free_host(indices_host);
        free_host(bounds_host);

        mesh.bvh = bvh_device;
    }

    Mesh* mesh_device = (Mesh*)alloc_device(sizeof(Mesh));
    memcpy_h2d(mesh_device, &mesh, sizeof(Mesh));
    
    // save descriptor
    uint64_t mesh_id = (uint64_t)mesh_device;
    add_descriptor(mesh_id, mesh);

    return mesh_id;
}

void mesh_destroy_host(uint64_t id)
{
    Mesh* m = (Mesh*)(id);

    delete[] m->bounds;
    bvh_destroy_host(m->bvh);

    delete m;
}

void mesh_destroy_device(uint64_t id)
{
    Mesh mesh;
    if (get_descriptor(id, mesh))
    {    
        bvh_destroy_device(mesh.bvh);
        free_device(mesh.bounds);
        free_device((Mesh*)id);

        rem_descriptor(id);
    }
}

void mesh_refit_host(uint64_t id)
{
    Mesh* m = (Mesh*)(id);

    for (int i=0; i < m->num_tris; ++i)
    {
        m->bounds[i] = bounds3();
        m->bounds[i].add_point(m->points[m->indices[i*3+0]]);
        m->bounds[i].add_point(m->points[m->indices[i*3+1]]);
        m->bounds[i].add_point(m->points[m->indices[i*3+2]]);
    }

    bvh_refit_host(m->bvh, m->bounds);
}


__global__ void compute_triangle_bounds(int n, const vec3* points, const int* indices, bounds3* b)
{
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;

    if (tid < n)
    {
        // if leaf then update bounds
        int i = indices[tid*3+0];
        int j = indices[tid*3+1];
        int k = indices[tid*3+2];

        vec3 p = points[i];
        vec3 q = points[j];
        vec3 r = points[k];

        vec3 lower = min(min(p, q), r);
        vec3 upper = max(max(p, q), r);

        b[tid] = bounds3(lower, upper);
    }
}


void mesh_refit_device(uint64_t id)
{
    // recompute triangle bounds
    Mesh m;
    if (get_descriptor(id, m))
    {
        const int num_threads_per_block = 256;
        const int num_blocks = (m.num_tris + num_threads_per_block - 1)/num_threads_per_block;

        compute_triangle_bounds<<<num_blocks, num_threads_per_block>>>(m.num_tris, m.points, m.indices, m.bounds);

        bvh_refit_device(m.bvh, m.bounds);
    }
}

#include "hip/hip_runtime.h"
#include "warp.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <string>

#if defined(__linux__)
#include <dlfcn.h>
static void* GetProcAddress(void* handle, const char* name) { return dlsym(handle, name); }
#endif

#if defined(_WIN32)
#include <windows.h>
#endif

#include "hip/hiprtc.h"

typedef hipError_t CUDAAPI cuInit_t(unsigned int);
typedef hipError_t CUDAAPI cuDeviceGet_t(hipDevice_t *dev, int ordinal);
typedef hipError_t CUDAAPI cuCtxGetCurrent_t(hipCtx_t* ctx);
typedef hipError_t CUDAAPI cuCtxSetCurrent_t(hipCtx_t ctx);
typedef hipError_t CUDAAPI cuCtxCreate_t(hipCtx_t* pctx, unsigned int flags, hipDevice_t dev);
typedef hipError_t CUDAAPI cuCtxDestroy_t(hipCtx_t pctx);

static cuInit_t* cuInit_f;
static cuCtxGetCurrent_t* cuCtxGetCurrent_f;
static cuCtxSetCurrent_t* cuCtxSetCurrent_f;
//static cuCtxCreate_t* cuCtxCreate_f;
//static cuCtxDestroy_t* cuCtxDestroy_f;
//static cuDeviceGet_t* cuDeviceGet_f;

static hipCtx_t g_cuda_context;
static hipCtx_t g_save_context;

static hipStream_t g_cuda_stream;

int cuda_init()
{
    #if defined(_WIN32)
        static HMODULE hCudaDriver = LoadLibrary("nvcuda.dll");
    #elif defined(__linux__)
        static void* hCudaDriver = dlopen("libcuda.so", RTLD_NOW);
    #endif

    if (hCudaDriver == NULL)
        return false;

	cuInit_f = (cuInit_t*)GetProcAddress(hCudaDriver, "hipInit");
	cuCtxSetCurrent_f = (cuCtxSetCurrent_t*)GetProcAddress(hCudaDriver, "hipCtxSetCurrent");
	cuCtxGetCurrent_f = (cuCtxGetCurrent_t*)GetProcAddress(hCudaDriver, "hipCtxGetCurrent");

    if (cuInit_f == NULL)
        return -1;

    hipError_t err = cuInit_f(0);    
    if (err != hipSuccess)
		return err;

    hipCtx_t ctx;
    cuCtxGetCurrent_f(&ctx);

    if (ctx == NULL)
    {
        // create a new default runtime context
        hipSetDevice(0);
        cuCtxGetCurrent_f(&ctx);
    }
    
    // save the context, all API calls must have this context set on the calling thread
    g_cuda_context = ctx;
    
    check_cuda(hipStreamCreate(&g_cuda_stream));
    
    return 0;
}

void* alloc_host(size_t s)
{
    void* ptr;
    check_cuda(hipHostMalloc(&ptr, s));
    return ptr;
}

void free_host(void* ptr)
{
    hipHostFree(ptr);
}

void* alloc_device(size_t s)
{
    void* ptr;
    check_cuda(hipMalloc(&ptr, s));

    return ptr;
}

void free_device(void* ptr)
{
    check_cuda(hipFree(ptr));
}

void memcpy_h2d(void* dest, void* src, size_t n)
{
    check_cuda(hipMemcpyAsync(dest, src, n, hipMemcpyHostToDevice, g_cuda_stream));
}

void memcpy_d2h(void* dest, void* src, size_t n)
{
    check_cuda(hipMemcpyAsync(dest, src, n, hipMemcpyDeviceToHost, g_cuda_stream));
}

void memcpy_d2d(void* dest, void* src, size_t n)
{
    check_cuda(hipMemcpyAsync(dest, src, n, hipMemcpyDeviceToDevice, g_cuda_stream));
}

void memset_device(void* dest, int value, size_t n)
{
    check_cuda(hipMemsetAsync(dest, value, n, g_cuda_stream));
}

void synchronize()
{
    check_cuda(hipStreamSynchronize(g_cuda_stream));
}

void array_inner_device(uint64_t a, uint64_t b, uint64_t out, int len)
{

}

void array_sum_device(uint64_t a, uint64_t out, int len)
{
    
}


uint64_t cuda_check_device()
{
    hipDeviceSynchronize();
    return hipPeekAtLastError(); 
}

void cuda_report_error(int code, const char* file, int line)
{
    if (code != hipSuccess) 
    {
        printf("CUDA Error: %s %s %d\n", hipGetErrorString((hipError_t)code), file, line);
    }
}

void* cuda_get_stream()
{
    return g_cuda_stream;
}

void cuda_graph_begin_capture()
{
    check_cuda(hipStreamBeginCapture(g_cuda_stream, hipStreamCaptureModeGlobal));
}

void* cuda_graph_end_capture()
{
    hipGraph_t graph;
    check_cuda(hipStreamEndCapture(g_cuda_stream, &graph));

    hipGraphExec_t graph_exec;
    check_cuda(hipGraphInstantiate(&graph_exec, graph, NULL, NULL, 0))

    // free source graph
    check_cuda(hipGraphDestroy(graph));

    return graph_exec;
}

void cuda_graph_launch(void* graph_exec)
{
    check_cuda(hipGraphLaunch((hipGraphExec_t)graph_exec, g_cuda_stream));
}

void cuda_graph_destroy(void* graph_exec)
{
    check_cuda(hipGraphExecDestroy((hipGraphExec_t)graph_exec));
}

void cuda_acquire_context()
{
    cuCtxGetCurrent_f(&g_save_context);
    cuCtxSetCurrent_f(g_cuda_context);
}

void cuda_restore_context()
{
    cuCtxSetCurrent_f(g_save_context);
}


void* cuda_get_context()
{
	return g_cuda_context;
}

void cuda_set_context(void* ctx)
{
    g_cuda_context = (hipCtx_t)ctx;
}

const char* cuda_get_device_name()
{
    static hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    return prop.name;
}

size_t cuda_compile_program(const char* cuda_src, const char* include_dir, bool debug, bool verbose, const char* output_file)
{
    hiprtcResult res;

    hiprtcProgram prog;
    res = hiprtcCreateProgram(
        &prog,          // prog
        cuda_src,      // buffer
        NULL,          // name
        0,             // numHeaders
        NULL,          // headers
        NULL);         // includeNames

    if (res != HIPRTC_SUCCESS)
        return res;

    std::string include_opt = std::string("--include-path=") + include_dir;

    const char *opts[] = 
    {   
        "--device-as-default-execution-space",
        "--gpu-architecture=sm_35",
//        "--use_fast_math",
        "--std=c++11",
        "--define-macro=WP_CUDA",
        "--define-macro=WP_NO_CRT",
        "--define-macro=NDEBUG",
        include_opt.c_str()
    };

    res = hiprtcCompileProgram(prog, 7, opts);

    if (res == HIPRTC_SUCCESS)
    {
        // save ptx
        size_t ptx_size;
        hiprtcGetCodeSize(prog, &ptx_size);

        char* ptx = (char*)malloc(ptx_size);
        hiprtcGetCode(prog, ptx);

        // write to file
        FILE* file = fopen(output_file, "w");
        fwrite(ptx, 1, ptx_size, file);
        fclose(file);

        free(ptx);
    }

    if (res != HIPRTC_SUCCESS || verbose)
    {
        // get program log
        size_t log_size;
        hiprtcGetProgramLogSize(prog, &log_size);

        char* log = (char*)malloc(log_size);
        hiprtcGetProgramLog(prog, log);

        // todo: figure out better way to return this to python
        printf(log);
        free(log);
    }

    hiprtcDestroyProgram(&prog);
    return res;
}

void* cuda_load_module(const char* path)
{
    FILE* file = fopen(path, "r");
    fseek(file, 0, SEEK_END);
    size_t length = ftell(file);
    fseek(file, 0, SEEK_SET);

    char* buf = (char*)malloc(length);
    fread(buf, length, 1, file);
    fclose(file);

    hipModule_t module = NULL;
    hipError_t res = hipModuleLoadDataEx(&module, buf, 0, 0, 0);
    if (res != HIPRTC_SUCCESS)
        printf("Warp: Loading PTX module failed with error: %d\n", res);

    free(buf);

    return module;
}

void cuda_unload_module(void* module)
{
    hipModuleUnload((hipModule_t)module);
}

void* cuda_get_kernel(void* module, const char* name)
{
    hipFunction_t kernel = NULL;
    hipError_t res = hipModuleGetFunction(&kernel, (hipModule_t)module, name);
    if (res != HIPRTC_SUCCESS)
        printf("Warp: Failed to lookup kernel function %s in module\n", name);

    return kernel;
}

size_t cuda_launch_kernel(void* kernel, int dim, void** args)
{
    const int block_dim = 256;
    const int grid_dim = (dim + block_dim - 1)/block_dim;

    hipError_t res = hipModuleLaunchKernel(
        (hipFunction_t)kernel,
        grid_dim, 1, 1,
        block_dim, 1, 1,
        0, g_cuda_stream,
        args,
        0);

    return res;

}

// impl. files
#include "bvh.cu"
#include "mesh.cu"
//#include "spline.inl"
//#include "volume.inl"


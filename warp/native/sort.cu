/** Copyright (c) 2022 NVIDIA CORPORATION.  All rights reserved.
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

#include "warp.h"
#include "sort.h"

#define THRUST_IGNORE_CUB_VERSION_CHECK

#include <hipcub/hipcub.hpp>

#include <map>

// temporary buffer for radix sort
struct RadixSortTemp
{
    void* mem = NULL;
    size_t size = 0;
};

// map temp buffers to CUDA contexts
static std::map<hipCtx_t, RadixSortTemp> g_radix_sort_temp_map;

void radix_sort_reserve(int n, void** mem_out, size_t* size_out)
{
    hipcub::DoubleBuffer<int> d_keys;
	hipcub::DoubleBuffer<int> d_values;

    // compute temporary memory required
	size_t sort_temp_size;
	hipcub::DeviceRadixSort::SortPairs(NULL, sort_temp_size, d_keys, d_values, int(n), 0, 32, (hipStream_t)cuda_stream_get_current());

    hipCtx_t ctx;
    check_cu(cuCtxGetCurrent_f(&ctx));

    RadixSortTemp& temp = g_radix_sort_temp_map[ctx];

    if (sort_temp_size > temp.size)
    {
	    free_device(temp.mem);
        temp.mem = alloc_device(sort_temp_size);
        temp.size = sort_temp_size;
    }
    
    if (mem_out)
        *mem_out = temp.mem;
    if (size_out)
        *size_out = temp.size;
}

void radix_sort_pairs_device(int* keys, int* values, int n)
{
    hipcub::DoubleBuffer<int> d_keys(keys, keys + n);
	hipcub::DoubleBuffer<int> d_values(values, values + n);

    RadixSortTemp temp;
    radix_sort_reserve(n, &temp.mem, &temp.size);

    // sort
    hipcub::DeviceRadixSort::SortPairs(
        temp.mem,
        temp.size,
        d_keys, 
        d_values, 
        n, 0, 32, 
        (hipStream_t)cuda_stream_get_current());

	if (d_keys.Current() != keys)
		memcpy_d2d(keys, d_keys.Current(), sizeof(int)*n);

	if (d_values.Current() != values)
		memcpy_d2d(values, d_values.Current(), sizeof(int)*n);
}